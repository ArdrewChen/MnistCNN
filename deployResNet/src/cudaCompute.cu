
#include <hip/hip_runtime.h>
// 一些用于计算的cuda函数

// 从主机端复制数据至设备端
void copyInputToDevice(int input_data_size, int output_data_size)
{
    float *device_input_data = nullptr;
    float *device_output_data = nullptr;
    hipMalloc((void **)&device_input_data, input_data_size);
    hipMalloc((void **)device_output_data, output_data_size);
}
